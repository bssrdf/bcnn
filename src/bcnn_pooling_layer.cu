
#include <hip/hip_runtime.h>
/*
* Copyright (c) 2016 Jean-Noel Braun.
*
* Permission is hereby granted, free of charge, to any person obtaining a copy
* of this software and associated documentation files (the "Software"), to deal
* in the Software without restriction, including without limitation the rights
* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
* copies of the Software, and to permit persons to whom the Software is
* furnished to do so, subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in
* all copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
* SOFTWARE.
*/


#ifdef BCNN_USE_CUDA

#include "bcnn_pooling_layer.h"
#include "bcnn_utils.h"

__global__ void _bcnn_forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, float *input, float *output, int *indexes)
{
    int h = (in_h-1)/stride + 1;
    int w = (in_w-1)/stride + 1;
    int c = in_c;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) {
        return;
    }

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int out_index = j + w*(i + h*(k + c*b));
    float max = -INFINITY;
    int max_i = -1;
    int l, m;
    for (l = 0; l < size; ++l) {
        for (m = 0; m < size; ++m) {
            int cur_h = i * stride + l;
            int cur_w = j * stride + m;
            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            int valid = (cur_h >= 0 && cur_h < in_h &&
                    cur_w >= 0 && cur_w < in_w);
            float val = (valid != 0) ? input[index] : -INFINITY;
            max_i = (val > max) ? index : max_i;
            max   = (val > max) ? val   : max;
        }
    }
    output[out_index] = max;
    indexes[out_index] = max_i;
}

int bcnn_forward_maxpool_layer_gpu(bcnn_layer *layer, bcnn_node *src_node, bcnn_node *dst_node)
{
    bcnn_tensor src = src_node->tensor;
    bcnn_tensor dst = dst_node->tensor;
/*#ifdef BCNN_USE_CUDNN
    float zero = 0.0f, one = 1.0f;
    bcnn_cudnn_check(cudnnPoolingForward(bcnn_cudnn_handle(), layer->pooling_desc,
        &one, layer->src_tensor_desc, src.data_gpu, &zero,
        layer->dst_tensor_desc, dst.data_gpu));
#else*/
    int sz = bcnn_tensor_get_size(&dst);

    _bcnn_forward_maxpool_layer_kernel<<<bcnn_cuda_gridsize(sz), BCNN_CUDA_THREADS>>>(sz,
        src.w, src.h, src.c, layer->stride, layer->size, src.data_gpu, dst.data_gpu, layer->indexes_gpu);
    bcnn_cuda_check(cudaPeekAtLastError());
//#endif

    return BCNN_SUCCESS;
}

__global__ void _bcnn_backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size,
     float *diff, float *prev_delta, int *indexes)
{
    int h = (in_h-1)/stride + 1;
    int w = (in_w-1)/stride + 1;
    int c = in_c;
    int area = (size-1)/stride;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) {
        return;
    }

    int index = id;
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    int w_offset = (-size-1)/2 + 1;
    int h_offset = (-size-1)/2 + 1;

    float d = 0;
    int l, m;
    for (l = -area; l < area + 1; ++l) {
        for (m = -area; m < area + 1; ++m) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;
            int out_index = out_w + w * (out_h + h * (k + c * b));
            int valid = (out_w >= 0 && out_w < w &&
                     out_h >= 0 && out_h < h);
            d += (valid && indexes[out_index] == index) ? diff[out_index] : 0;
        }
    }
    prev_delta[index] += d;
}

int bcnn_backward_maxpool_layer_gpu(bcnn_layer *layer, bcnn_node *src_node, bcnn_node *dst_node)
{
    bcnn_tensor src = src_node->tensor;
    bcnn_tensor dst = dst_node->tensor;
/*#ifdef BCNN_USE_CUDNN
    float zero = 0.0f, one = 1.0f;
    bcnn_cudnn_check(cudnnPoolingBackward(bcnn_cudnn_handle(), layer->pooling_desc,
        &one, layer->dst_tensor_desc, dst.data_gpu, layer->dst_tensor_desc, dst.grad_data_gpu,
        layer->src_tensor_desc, src.data_gpu, &zero, layer->src_tensor_desc, src.grad_data_gpu));
#else*/
    int sz = bcnn_tensor_get_size(&src);

    _bcnn_backward_maxpool_layer_kernel<<<bcnn_cuda_gridsize(sz), BCNN_CUDA_THREADS>>>(sz,
        src.w, src.h, src.c, layer->stride, layer->size, dst.grad_data_gpu, src.grad_data_gpu, layer->indexes_gpu);
    bcnn_cuda_check(cudaPeekAtLastError());
//#endif

    return BCNN_SUCCESS;
}

#endif